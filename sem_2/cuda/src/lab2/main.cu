#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define CSC(call)                                                    \
do {                                                                \
    hipError_t res = call;                                            \
    if (res != hipSuccess) {                                        \
        fprintf(stderr, "ERROR in %s:%d. Message: %s\n",            \
                __FILE__, __LINE__, hipGetErrorString(res));        \
        exit(0);                                                    \
    }                                                                \
} while(0)

// текстурная ссылка <тип элементов, размерность, режим нормализации>
texture<uchar4, 2, hipReadModeElementType> tex;

__global__ void kernel(uchar4 *out, int w, int h, int delta_w, int delta_h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int x, y;
    for (y = idy; y < (h/delta_h); y += offsety) {
        for (x = idx; x < (w/delta_w); x += offsetx) {
            int xx = 0;
            int yy = 0;
            int zz = 0;
            int ww = 0;
            for (int inner_x = x*delta_w; inner_x < x*delta_w + delta_w; inner_x++) {
                for (int inner_y = y*delta_h; inner_y < y*delta_h + delta_h; inner_y++) {
                    xx += (tex2D(tex, inner_x, inner_y)).x;
                    yy += (tex2D(tex, inner_x, inner_y)).y;
                    zz += (tex2D(tex, inner_x, inner_y)).z;
                    ww += (tex2D(tex, inner_x, inner_y)).w;
                }
            }
            out[y*(w/delta_w) + x] = make_uchar4(
                    xx/(delta_h*delta_w), yy/(delta_h*delta_w), , ww/(delta_h*delta_w)
            );
        }
    }
}

int main() {
    int w, h;
//    char in[9999];  // G:\Projects\CUDA\lab2\original.bin
//    scanf("%s", in);
//    char out[9999];  // G:\Projects\CUDA\lab2\result.bin
//    scanf("%s", out);
    int w_new = 335;
    int h_new = 93;
//    scanf("%d", &w_new);
//    scanf("%d", &h_new);
//    int delta_w = 67;  // 1, 3, 5, 15, 67,  201,  335, 1 005
//    int delta_h = 31;  // 1, 2, 3, 6, 9, 18, 31, 62, 93,  186,  279,  558
    FILE *fp = fopen("G:\\Projects\\CUDA\\lab2\\original.bin", "rb");
    fread(&w, sizeof(int), 1, fp);
    fread(&h, sizeof(int), 1, fp);
    int delta_w = w / w_new;
    int delta_h = h / h_new;
    uchar4 *data = (uchar4 *) malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);
    // Подготовка данных для текстуры
    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * (w) * (h), hipMemcpyHostToDevice));

    // Подготовка текстурной ссылки, настройка интерфейса работы с данными
    tex.addressMode[0] = hipAddressModeClamp;    // Политика обработки выхода за границы по каждому измерению
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = ch;
    tex.filterMode = hipFilterModePoint;        // Без интерполяции при обращении по дробным координатам
    tex.normalized = false;                        // Режим нормализации координат: без нормализации

    // Связываем интерфейс с данными
    CSC(hipBindTextureToArray(tex, arr, ch));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * (w / delta_w) * (h / delta_h)));





    hipEvent_t start, stop;
    float t;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));
    CSC(hipEventRecord(start, 0));
    kernel<<<dim3(512, 512), dim3(16, 16)>>>(dev_out, w, h, delta_w, delta_h);
    CSC(hipGetLastError());
    CSC(hipEventRecord(stop, 0));
    CSC(hipEventSynchronize(stop));
    CSC(hipEventElapsedTime(&t, start, stop));
    printf("time = %f\n", t);
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));




    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * (w / delta_w) * (h / delta_h), hipMemcpyDeviceToHost));

    // Отвязываем данные от текстурной ссылки
    CSC(hipUnbindTexture(tex));

    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    int a1 = (w / delta_w);
    int a2 = (h / delta_h);
    fp = fopen("G:\\\\Projects\\\\CUDA\\\\lab2\\\\result.bin", "wb");
    fwrite(&a1, sizeof(int), 1, fp);
    fwrite(&a2, sizeof(int), 1, fp);
    fwrite(data, sizeof(uchar4), (w / delta_w) * (h / delta_h), fp);
    fclose(fp);

    free(data);
    return 0;
}
